// Vector Addition

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void vectorAdd(int* a, int* b, int* c, int n)
{
    //Calculate Index Thread
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    //Make sure we stay in bounds
    if(tid<n)
        //Vector Addition      
        c[tid] = a[tid] + b[tid];
}

int main()
{
    //Number of elements
    int n = 16;

    //Host Pointers
    int* h_a;
    int* h_b;
    int* h_c;

    //Device Pointers
    int* d_a;
    int* d_b;
    int* d_c;

    size_t bytes = n * sizeof(int);

    //Allocating memory on Host side
    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);

    //Initializing host vectors
    for(int i=0;i<n;i++)
    {
        h_a[i]=1;
        h_b[i]=2;
    }

    printf("Matrix A: \n");
    for(int i=0;i<n;i++)
    {
        printf("%d ",h_a[i]);
    }

    printf("\nMatrix B: \n");
    for(int i=0;i<n;i++)
    {
        printf("%d ",h_b[i]);
    }

    //Allocating memory on Device side
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    //Init block and grid size
    int block_size = 4;
    int grid_size = (int)ceil( (float) n / block_size);
    printf("Grid size is %d\n",grid_size);

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    
    vectorAdd<<< grid_size, block_size >>>(d_a, d_b, d_c, n);

    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    printf("Matrix C: \n");
    for(int i=0;i<n;i++)
    {
        printf("%d ",h_c[i]);
    }

    printf("Completed Successfully!\n");

    //Clean-Up
    free(h_a);
    free(h_b);
    free(h_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}